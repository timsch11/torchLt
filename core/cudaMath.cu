#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <string>
#include "util.cu"


// MATH FUNCTIONS

// adds one entry each
__global__ void addVecEntries(float* d_targetMemorySpace, float* d_vec1, float* d_vec2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_targetMemorySpace[idx] = d_vec1[idx] + d_vec2[idx];
}

// adds the values of two vectors and stores the result in <targetMemorySpace>, Note: for efficiency reasons size of targetMemorySpace must be rounded up to a multiple of blocksize
hipError_t vecadd(float* d_targetMemorySpace, float* d_vector1, unsigned int vectorSize1, float* d_vector2, unsigned int vectorSize2) {

    // check for vector compatibility
    if (vectorSize1 != vectorSize2) {
        printf("vectors to be added have different shapes\n");
        return hipErrorInvalidValue;
    }

    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(vectorSize1);
    
    addVecEntries<<<blocksThreads.first, blocksThreads.second, 0, 0>>>(d_targetMemorySpace, d_vector1, d_vector2);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    return hipSuccess;
}

// adds one column each
__global__ void subtractVecEntries(float* d_targetMemorySpace, float* d_vec1, float* d_vec2) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    d_targetMemorySpace[ind] = d_vec1[ind] - d_vec2[ind];
}

// subtracts the values of two vectors and stores the result in <targetMemorySpace>, Note: for efficiency reasons size of targetMemorySpace must be rounded up to a multiple of blocksize
hipError_t vecsub(float* d_targetMemorySpace, float* d_vector1, unsigned int vectorSize1, float* d_vector2, unsigned int vectorSize2) {

    // check for vector compatibility
    if (vectorSize1 != vectorSize2) {
        printf("vectors to be subtracted have different shapes\n");
        return hipErrorInvalidValue;
    }

    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(vectorSize1);
    
    subtractVecEntries<<<blocksThreads.first, blocksThreads.second, 0, 0>>>(d_targetMemorySpace, d_vector1, d_vector2);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipGetLastError());
    
    return hipSuccess;
}

__global__ void scaleEntries(float* d_targetMemorySpace, float* d_tensor, float scalar) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_targetMemorySpace[idx] = d_tensor[idx] * scalar;
}

// performs scalar multiplication on the elements of an arbitrarily shaped tensor
hipError_t scaletensor(float* d_targetMemorySpace, float* d_tensor, unsigned int tensorSize, float scalar) {

    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(tensorSize);
    
    scaleEntries<<<blocksThreads.first, blocksThreads.second, 0, 0>>>(d_targetMemorySpace, d_tensor, scalar);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    return hipSuccess;
}

__global__ void hadamard_kernel(float* d_targetMemorySpace, float* d_tensor1, float* d_tensor2) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    d_targetMemorySpace[ind] = d_tensor1[ind] * d_tensor2[ind];
}

// computes the hadamard product between tensor1 and tensor2 and stores the result in targetMemorySpace, the tensor shapes must obviously match
hipError_t hadamard(float* d_targetMemorySpace, float* d_tensor1, float* d_tensor2, std::pair<unsigned int, unsigned int> shape) {

    // calculate num of fields
    unsigned int size = (shape.second == 0) ? shape.first : shape.first * shape.second;

    // calculate #Block and #Thread
    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(size);
    
    // let kernel do its work
    hadamard_kernel<<<blocksThreads.first, blocksThreads.second, 0, 0>>>(d_targetMemorySpace, d_tensor1, d_tensor2);

    // error checking
    CHECK_CUDA_ERROR(hipGetLastError());

    return hipSuccess;
}


/*int main() {
    // Allocate host memory for results
float* h_bias = new float[256];  // Host memory

// Device allocation
float* d_bias = constants(25, 1);
// hipMalloc(&d_bias, 5 * sizeof(float));

// Initialize with zeros


float* d_bias2 = constants(25, -2);

// Initialize with zeros
// weight_init(d_bias, 5, 1, 10.75f, 1234567);

float* d_target = zeros(25);

hipEvent_t e1, e2;
hipEventCreate(&e1);
hipEventCreate(&e2);

hipStream_t s1;
hipStreamCreate(&s1);


hipEventRecord(e1, s1);
matmatT_matmul<<<5, 5, 5*sizeof(float), s1>>>(d_bias, d_bias2, 5, d_target);
hipEventRecord(e2, s1);
CHECK_CUDA_ERROR(hipGetLastError());
hipDeviceSynchronize();

float milliseconds = 0;
hipEventElapsedTime(&milliseconds, e1, e2);

std::cout << milliseconds << "ms passed";

// Copy from device (zero_init) to device (d_bias)
// hipMemcpy(d_bias, zero_init, 5 * sizeof(float), hipMemcpyDeviceToDevice);

// Copy from device to host for printing
hipMemcpy(h_bias, d_target, 256 * sizeof(float), hipMemcpyDeviceToHost);
hipDeviceSynchronize();

// Check for errors
hipError_t error = hipGetLastError();
if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    throw std::runtime_error("CUDA error");
}

// Print values from host memory
for (int i = 0; i < 256; i++) {
    std::cout << h_bias[i] << " ";
}

// Cleanup
delete[] h_bias;
hipFree(d_bias);
hipFree(d_bias2);
}*/