#include "hip/hip_runtime.h"
#include "cudaMem.cuh"


float* reserveMemoryOnDevice(unsigned int size) {
    unsigned int allocationSize = size + BLOCK_SIZE - (size % BLOCK_SIZE);

    // declare pointer
    float* memoryAlloc = nullptr;

    // reserve actual space in memory, add some padding for thread efficiency
    hipError_t allocStatus = hipMalloc(&memoryAlloc, 
        (size + BLOCK_SIZE - (size % BLOCK_SIZE)) * sizeof(float));
    
    if (allocStatus != hipSuccess || memoryAlloc == nullptr) {
        std::cout << std::string(hipGetErrorString(allocStatus));
        throw std::runtime_error("CUDA memory allocation failed: " + 
            std::string(hipGetErrorString(allocStatus)));
    }

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    return memoryAlloc;
}

__global__ void __initMemCell(float* d_memorySection, float value) {
    d_memorySection[blockIdx.x * blockDim.x + threadIdx.x] = value;
}

float* zeros(unsigned int size) {
    // returns a pointer to (first element of) an array (interpretation of dimension is up to the caller) of specified size filled with zeros; array lives in unified memory (on cpu and gpu)

    // calc block/thread allocation scheme
    std::pair<unsigned int, unsigned int> blockThreadAllocation = computeBlockThreadAllocation(size);

    // reserve memory
    float* d_memoryAllocation = reserveMemoryOnDevice(blockThreadAllocation.first * blockThreadAllocation.second);

    // launch kernel
    __initMemCell<<<blockThreadAllocation.first, blockThreadAllocation.second, 0, 0>>>(d_memoryAllocation, 0.0f);

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    CHECK_CUDA_ERROR(hipGetLastError());

    return d_memoryAllocation;
}

float* constants(unsigned int size, float constant) {

    // calc block/thread allocation scheme
    std::pair<unsigned int, unsigned int> blockThreadAllocation = computeBlockThreadAllocation(size);

    // reserve memory
    float* d_memoryAllocation = reserveMemoryOnDevice(size);

    // launch kernel
    __initMemCell<<<blockThreadAllocation.first, blockThreadAllocation.second, 0, 0>>>(d_memoryAllocation, constant);

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    CHECK_CUDA_ERROR(hipGetLastError());

    return d_memoryAllocation;
}

void constants(float* d_value, unsigned int size, float constant) {

    // calc block/thread allocation scheme
    std::pair<unsigned int, unsigned int> blockThreadAllocation = computeBlockThreadAllocation(size);

    // launch kernel
    __initMemCell<<<blockThreadAllocation.first, blockThreadAllocation.second, 0, 0>>>(d_value, constant);

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    CHECK_CUDA_ERROR(hipGetLastError()); 
}

__global__ void __transposeMemDup(float* d_source, float* d_destination, int size) {
    int ind = size - blockIdx.x * blockDim.x + threadIdx.x;
    if (size >= 0) {
        d_destination[ind] = d_source[ind];
    }
}

__global__ void __memDup(float* d_source, float* d_destination) {
    // calculate index, block is responsible for arr[n] to arr[n+blockSize] elements to leverage coalescing access
    unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
    d_destination[ind] = d_source[ind];
}

void cudaMemDup(float* d_source, float* d_destination, unsigned int size, bool transpose) {
    // calc block/thread allocation scheme
    std::pair<unsigned int, unsigned int> blockThreadAllocation = computeBlockThreadAllocation(size);

    // select which kernel to use for copying
    if (transpose) {
        __transposeMemDup<<<blockThreadAllocation.first, blockThreadAllocation.second, 0, 0>>>(d_source, d_destination, size);
    } else {
        __memDup<<<blockThreadAllocation.first, blockThreadAllocation.second, 0, 0>>>(d_source, d_destination);
    }

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // error checking 
    CHECK_CUDA_ERROR(hipGetLastError());
}

// WEIGHT INITIALIZATION FUNCTIONS

__global__ void __cuda_weight_init(float* weights, unsigned int size, float scalingFactor, int seed) {
    // declaring random state
    hiprandState state;

    // set index
    int ind = blockDim.x * blockIdx.x + threadIdx.x;

    // init hiprand
    hiprand_init(seed + ind, blockIdx.x, 0, &state);

    // set weight
    weights[ind] = hiprand_normal(&state) * sqrtf(scalingFactor);
}

void weight_init(float* d_targetMemorySpace, unsigned int size, float scaling_factor, int seed) {

    // add some padding to ensure that kernel runs efficiently but also does not override other memory cells
    std::pair<unsigned int, unsigned int> blockThreadAllocation = computeBlockThreadAllocation(size);

    // run kernel
    __cuda_weight_init<<<blockThreadAllocation.first, blockThreadAllocation.second, 0, 0>>>(d_targetMemorySpace, size, scaling_factor, seed);
    
    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    CHECK_CUDA_ERROR(hipGetLastError());
}
