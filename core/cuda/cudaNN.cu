#include "hip/hip_runtime.h"
#include "cudaNN.cuh"


// ACTIVATION FUNCTIONS


__global__ void __relu(float* d_targetMemorySpace, float* vector) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (vector[i] > 0) {
        d_targetMemorySpace[i] = vector[i];
    } else {
        d_targetMemorySpace[i] = 0;
    }
}

hipError_t relu(float* d_targetMemorySpace, float* d_vector, unsigned int size) {
    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(size);
    __relu<<<blocksThreads.first, blocksThreads.second>>>(d_targetMemorySpace, d_vector);

    hipError_t err = hipGetLastError();
    
    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    return err;
}

float* reluAlloc(float* d_vector, unsigned int size) {
    float* result = reserveMemoryOnDevice(size);

    if (result == nullptr) {
        return nullptr;
    }

    hipError_t err = relu(result, d_vector, size);

    if (err != hipSuccess) {
        std::cout << "Cuda error when performing relu: " << std::string(hipGetErrorString(err));
        return nullptr;
    }

    return result;
}

__global__ void __sigmoid(float* d_targetMemorySpace, float* d_tensor) {
    unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
    d_targetMemorySpace[ind] = 1.0f / (1.0f + expf(-d_tensor[ind]));
}

hipError_t sigmoid(float* d_targetMemorySpace, float* d_tensor, unsigned int size) {
    // compute optimal block/thread distribution
    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(size);

    // execute computation
    __sigmoid<<<blocksThreads.first, blocksThreads.second>>>(d_targetMemorySpace, d_tensor);

    hipError_t err = hipGetLastError();
    
    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    return err;
}

float* sigmoidAlloc(float* d_tensor, unsigned int size) {
    // allocate required memory (+padding)
    float* d_result = reserveMemoryOnDevice(size);

    if (d_result == nullptr) {
        return nullptr;
    }

    // check for errors
    hipError_t err = sigmoid(d_result, d_tensor, size);


    if (err != hipSuccess) {
        std::cout << "Cuda error when performing sigmoid: " << std::string(hipGetErrorString(err));
        return nullptr;
    }

    return d_result;
}

__global__ void __tanh(float* d_targetMemorySpace, float* d_tensor) {
    unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;
    d_targetMemorySpace[ind] = 1.0f - (2.0f / (expf(2.0f * d_tensor[ind]) + 1));
}

hipError_t tanh(float* d_targetMemorySpace, float* d_tensor, unsigned int size) {
    // compute optimal block/thread distribution
    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(size);

    // execute computation
    __tanh<<<blocksThreads.first, blocksThreads.second>>>(d_targetMemorySpace, d_tensor);

    hipError_t err = hipGetLastError();
    
    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    return err;
}

float* tanhAlloc(float* d_tensor, unsigned int size) {
    // allocate required memory (+padding)
    float* d_result = reserveMemoryOnDevice(size);

    if (d_result == nullptr) {
        return nullptr;
    }

    // check for errors
    hipError_t err = tanh(d_result, d_tensor, size);

    if (err != hipSuccess) {
        std::cout << "Cuda error when performing tanh: " << std::string(hipGetErrorString(err));
        return nullptr;
    }

    return d_result;
}

// WEIGHT INITIALIZATION


hipError_t kaiming_he(float* d_targetMemorySpace, unsigned int in_features, unsigned int out_features, int seed) {
    // set scaling factor for kaiming he init
    float scaling_factor = 2.0 / out_features;

    return weight_init(d_targetMemorySpace, in_features * out_features, scaling_factor, seed);
}

hipError_t xavier(float* d_targetMemorySpace, unsigned int in_features, unsigned int out_features, int seed) {
    // set scaling factor for xavier init
    float scaling_factor = 1.0 / out_features;

    return weight_init(d_targetMemorySpace, in_features * out_features, scaling_factor, seed);
}

// WEIGHT UPDATE

// updates the given weight matrix (passed as pointer to float array), performs error checking 
void updateWeightMatrix(float* d_weightMatrixToUpdate, float* d_gradient, unsigned int in_features, unsigned int out_features, float learningRate) {

}

// updates the given bias vector (passed as pointer to float array), performs error checking 
void updateBiasVector(float* d_biasVectorToUpdate, float* d_gradient, unsigned int out_features, float learningRate) {
    
}

// MATH


float* hadamardAlloc(float* d_tensor1, std::pair<unsigned int, unsigned int> shapeT1, float* d_tensor2, std::pair<unsigned int, unsigned int> shapeT2) {

    // check for compatibility
    if (shapeT1.first != shapeT2.first || shapeT1.second != shapeT2.second) {
        printf("incompatible shapes for hadamard product");
        return nullptr;
    }

    // allocate memory
    float* d_result = reserveMemoryOnDevice(shapeT1.first * shapeT1.second);

    if (d_result == nullptr) {
        printf("Error when allocating memory in hadamardAlloc");
        return nullptr;
    }

    // perform computation
    hipError_t err = hadamard(d_result, d_tensor1, d_tensor2, shapeT1);

    if (err != hipSuccess) {
        printf("Error when performing hadamard");
        return nullptr;
    }

    // return pointer to result
    return d_result;

} 

float* tensoraddAlloc(float* d_vector1, unsigned int vectorSize1, float* d_vector2, unsigned int vectorSize2) {
    // check for compatibility
    if (vectorSize1 != vectorSize2) {
        printf("incompatible shapes for vector addition");
        return nullptr;
    }

    // allocate memory
    float* d_result = reserveMemoryOnDevice(vectorSize1);

    if (d_result == nullptr) {
        printf("Error when allocating memory in hadamardAlloc");
        return nullptr;
    }

    // perform computation
    hipError_t err = tensoradd(d_result, d_vector1, vectorSize1, d_vector2, vectorSize2);

    if (err != hipSuccess) {
        printf("Error when performing add");
        return nullptr;
    }

    // return pointer to result
    return d_result;
}

float* tensorsubAlloc(float* d_vector1, unsigned int vectorSize1, float* d_vector2, unsigned int vectorSize2) {
    // check for compatibility
    if (vectorSize1 != vectorSize2) {
        printf("incompatible shapes for vector subtraction");
        return nullptr;
    }

    // allocate memory
    float* d_result = reserveMemoryOnDevice(vectorSize1);

    if (d_result == nullptr) {
        printf("Error when allocating memory in hadamardAlloc");
        return nullptr;
    }

    // perform computation
    hipError_t err = vecsub(d_result, d_vector1, vectorSize1, d_vector2, vectorSize2);

    if (err != hipSuccess) {
        printf("Error when performing sub");
        return nullptr;
    }

    // return pointer to result
    return d_result;
}

float* matmulAlloc(hipblasHandle_t* handle, int ax, int ay, int bx, int by, const float *A, const float *B) {
    if (ay != bx) {
        printf("invalid shapes for matrix multiplciation");
        return nullptr;
    }

    // allocate memory
    float* C = reserveMemoryOnDevice(ax * by);

    if (C == nullptr) {
        printf("Error when allocating memory for result of matmul");
        return nullptr;
    }
    
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasStatus_t matmulStatus;

    // call cuBLAS
    // c++ uses row major format, cuBLAS uses column major format
    // rowMajor(A) = columnMajor(A)T
    // this function essentially computes C = (BT AT)T
    matmulStatus = hipblasSgemm(*handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N, // No transpose for both A and B
                by, ax, bx,
                &alpha, B, by, // A is m x k
                A, bx, // B is k x n
                &beta, C, by); // C is m x n

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    if (matmulStatus != HIPBLAS_STATUS_SUCCESS) {
        hipFree(C);
        std::cout << "matrix multiplication failed: " << std::string(cublasGetStatusString(matmulStatus));
        return nullptr;
    }

    // return pointer to result
    return C;
}