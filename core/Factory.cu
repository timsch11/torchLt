#include "Tensor.h"


Tensor* createTensorFromInitFunction(std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, int seed, hipError_t(*initalization_function)(float*, unsigned int, unsigned int, int)) {
    return new Tensor(_shape, _track_gradient, seed, initalization_function);
}

Tensor* createTensorWithXavierInit(std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, int seed) {
    return new Tensor(_shape, _track_gradient, seed, &xavier);
}

Tensor* createTensorWithKaimingHeInit(std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, int seed) {
    return new Tensor(_shape, _track_gradient, seed, &kaiming_he);
}

Tensor* createTensorWithConstants(std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, float constant) {
    return new Tensor(_shape, _track_gradient, constant);
}

Tensor* createTensorFromDevice(float* _d_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, void (*_gradFunction)(Tensor*), Tensor* _d_funcArg1, std::pair<unsigned int, unsigned int> _shapeFuncArg1) {
    return new Tensor(_d_value, _shape, _track_gradient, _gradFunction, _d_funcArg1, _shapeFuncArg1);  
}

Tensor* createTensorFromDevice(float* _d_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, void (*_gradFunction)(Tensor*), Tensor* _d_funcArg1, std::pair<unsigned int, unsigned int> _shapeFuncArg1, Tensor* _d_funcArg2, std::pair<unsigned int, unsigned int> _shapeFuncArg2) {
    return new Tensor(_d_value, _shape, _track_gradient, _gradFunction, _d_funcArg1, _shapeFuncArg1, _d_funcArg2, _shapeFuncArg2);

}

Tensor* createTensorFromDevice(float* _d_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient) {
    return new Tensor(_d_value, _shape, _track_gradient);
}

Tensor* createTensorFromHost(float* _h_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient) {
    float* d_value;
    unsigned int size = _shape.first * _shape.second;

    hipError_t allocationError = hipMalloc(&d_value, size * sizeof(float));

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    if (allocationError != hipSuccess) {
        std::string errorString = "hipMalloc failed: " + std::string(hipGetErrorString(allocationError)) + "\n";
        std::cout << errorString;
        exit(EXIT_FAILURE);
    }

    hipError_t copyError = hipMemcpy(d_value, _h_value, size * sizeof(float), hipMemcpyHostToDevice);

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    if (copyError != hipSuccess) {
        std::string errorString = "cudaMemCpy failed: " + std::string(hipGetErrorString(copyError)) + "\n";
        hipFree(d_value);
        std::cout << errorString;
        exit(EXIT_FAILURE);
    }

    return new Tensor(d_value, _shape, _track_gradient);
}

void sync() {
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}


void init() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount == 0) {
        printf("No CUDA devices available");
        exit(EXIT_FAILURE);
    }

    err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("Failed to initialize CUDA device");
        exit(EXIT_FAILURE);
    }
}
