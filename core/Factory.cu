#include "Tensor.h"


Tensor* createTensorFromInitFunction(std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, int seed, hipError_t(*initalization_function)(float*, unsigned int, unsigned int, int)) {
    Tensor* obj = nullptr;

    try {
        obj = new Tensor(_shape, _track_gradient, seed, initalization_function);
    } catch(std::runtime_error exc) {
        delete obj;
        
        std::cout << "Error when trying to create Tensor: " << std::string(exc.what()) << "\n";
        exit(EXIT_FAILURE);
    }

    return obj;
}

Tensor* createTensorWithXavierInit(std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, int seed) {
    Tensor* obj = nullptr;

    try {
        obj = new Tensor(_shape, _track_gradient, seed, &xavier);
    } catch(std::runtime_error exc) {
        delete obj;
        
        std::cout << "Error when trying to create Tensor: " << std::string(exc.what()) << "\n";
        exit(EXIT_FAILURE);
    }

    return obj;
}

Tensor* createTensorWithKaimingHeInit(std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, int seed) {
    Tensor* obj = nullptr;

    try {
        obj = new Tensor(_shape, _track_gradient, seed, &kaiming_he);
    } catch(std::runtime_error exc) {
        delete obj;
        
        std::cout << "Error when trying to create Tensor: " << std::string(exc.what()) << "\n";
        exit(EXIT_FAILURE);
    }

    return obj;
}

Tensor* createTensorFromDevice(float* _d_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, void (*_gradFunction)(Tensor*), Tensor* _d_funcArg1, std::pair<unsigned int, unsigned int> _shapeFuncArg1) {
    Tensor* obj = nullptr;

    try {
        obj = new Tensor(_d_value, _shape, _track_gradient, _gradFunction, _d_funcArg1, _shapeFuncArg1);
    } catch(std::runtime_error exc) {
        delete obj;
        
        std::cout << "Error when trying to create Tensor: " << std::string(exc.what()) << "\n";
        exit(EXIT_FAILURE);
    }

    return obj;
}

Tensor* createTensorFromDevice(float* _d_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient, void (*_gradFunction)(Tensor*), Tensor* _d_funcArg1, std::pair<unsigned int, unsigned int> _shapeFuncArg1, Tensor* _d_funcArg2, std::pair<unsigned int, unsigned int> _shapeFuncArg2) {
    Tensor* obj = nullptr;

    try {
        obj = new Tensor(_d_value, _shape, _track_gradient, _gradFunction, _d_funcArg1, _shapeFuncArg1, _d_funcArg2, _shapeFuncArg2);
    } catch(std::runtime_error exc) {
        delete obj;

        std::cout << "Error when trying to create Tensor: " << std::string(exc.what()) << "\n";
        exit(EXIT_FAILURE);
    }

    return obj;
}

Tensor* createTensorFromDevice(float* _d_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient) {
    Tensor* obj = nullptr;

    try {
        obj = new Tensor(_d_value, _shape, _track_gradient);
    } catch(std::runtime_error exc) {
        delete obj;

        std::cout << "Error when trying to create Tensor: " << std::string(exc.what()) << "\n";
        exit(EXIT_FAILURE);
    }

    return obj;
}

Tensor* createTensorFromHost(float* _h_value, std::pair<unsigned int, unsigned int> _shape, bool _track_gradient) {
    float* d_value;
    unsigned int size = _shape.first * _shape.second;

    hipError_t allocationError = hipMalloc(&d_value, size * sizeof(float));

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    if (allocationError != hipSuccess) {
        std::string errorString = "hipMalloc failed: " + std::string(hipGetErrorString(allocationError)) + "\n";
        std::cout << errorString;
        exit(EXIT_FAILURE);
    }

    hipError_t copyError = hipMemcpy(d_value, _h_value, size * sizeof(float), hipMemcpyHostToDevice);

    // synchronize before continuing with host code
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    if (copyError != hipSuccess) {
        std::string errorString = "cudaMemCpy failed: " + std::string(hipGetErrorString(copyError)) + "\n";
        hipFree(d_value);
        std::cout << errorString;
        exit(EXIT_FAILURE);
    }

    Tensor* obj = nullptr;
    try {
        obj = new Tensor(d_value, _shape, _track_gradient);

        return obj;
    } catch(std::runtime_error exc) {
        hipFree(d_value);
        std::cout << "Error when trying to create Tensor: " << std::string(exc.what()) << "\n";
        exit(EXIT_FAILURE);
    }
}


void init() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount == 0) {
        printf("No CUDA devices available");
        exit(EXIT_FAILURE);
    }

    err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("Failed to initialize CUDA device");
        exit(EXIT_FAILURE);
    }
}
