#include "hip/hip_runtime.h"
#include "cudaMath.cu"
#include "cudaMem.cu"


/*All of the below implemented function return a pointer to the corresponding result location, this location is equal to the one passed as a parameter*/


// MEMORY 


// ACTIVATION FUNCTIONS

__global__ void relu_kernel(float* d_targetMemorySpace, float* vector) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (vector[i] > 0) {
        d_targetMemorySpace[i] = vector[i];
    } else {
        d_targetMemorySpace[i] = 0;
    }
}

// applies the relu activation function to a vector of arbitrary size (however only vectors are allowed no other type of tensors)
hipError_t relu(float* d_targetMemorySpace, float* d_vector, unsigned int size) {
    std::pair<unsigned int, unsigned int> blocksThreads = computeBlockThreadAllocation(size);
    relu_kernel<<<blocksThreads.first, blocksThreads.second>>>(d_targetMemorySpace, d_vector);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipGetLastError());
    return hipSuccess;
}


// DIFFERENTIATION OF ACTIVATION FUNCTIONS


// DIFFERENTIATION OF LOS FUNCTIONS

// WEIGHT INITIALIZATION

// applies the kaiming he inititalizaion to a memory location of sepcified size
void kaiming_he(float* d_targetMemorySpace, unsigned int in_features, unsigned int out_features, int seed) {
    // set scaling factor for kaiming he init
    float scaling_factor = 2.0 / out_features;

    weight_init(d_targetMemorySpace, in_features, out_features, scaling_factor, seed);
}

// applies the xavier inititalizaion to a memory location of sepcified size
void xavier(float* d_targetMemorySpace, unsigned int in_features, unsigned int out_features, int seed) {
    // set scaling factor for xavier init
    float scaling_factor = 1.0 / out_features;

    weight_init(d_targetMemorySpace, in_features, out_features, scaling_factor, seed);
}


// TODO
// FORWARD PASS

// performs a forward pass, stores result in specified memory locoation, assumes all tensors are in gpu memory, does error checking
void forward_layer(float* d_outputMemoryLocation, float* d_weights, float* d_bias, float* d_input, int inputSize, int in_features, int out_features) {
    try {

        // perform matrix multiplication with weights and input
        float* output = matvecmul(d_weights, out_features, in_features, d_input, inputSize);

        // add bias to result of matrix multiplication
        vecadd(output, inputSize, input, inputSize, output);

        return output;
    }
    catch(const std::runtime_error& e) {
        std::cerr << e.what() << '\n';
    }
}

// WEIGHT UPDATE

// updates the given weight matrix (passed as pointer to float array), performs error checking 
void updateWeightMatrix(float* d_weightMatrixToUpdate, float* d_gradient, unsigned int in_features, unsigned int out_features, float learningRate) {

}

// updates the given bias vector (passed as pointer to float array), performs error checking 
void updateBiasVector(float* d_biasVectorToUpdate, float* d_gradient, unsigned int out_features, float learningRate) {
    
}