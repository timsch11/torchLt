#include "hip/hip_runtime.h"
#include "util.cu"


// MEMORY INITIALIZATION FUNCTIONS

__global__ void initZero(float* d_memorySection) {
    d_memorySection[blockIdx.x * blockDim.x + threadIdx.x] = 0;
}

float* zeros(unsigned int size) {
    // returns a pointer to (first element of) an array (interpretation of dimension is up to the caller) of specified size filled with zeros; array lives in unified memory (on cpu and gpu)

    // calc block/thread allocation scheme
    std::pair<unsigned int, unsigned int> blockThreadAllocation = computeBlockThreadAllocation(size);

    // reserve memory
    float* d_memoryAllocation;
    CHECK_CUDA_ERROR(hipMalloc(&d_memoryAllocation, blockThreadAllocation.first * blockThreadAllocation.second * sizeof(float)));

    // launch kernel
    initZero<<<blockThreadAllocation.first, blockThreadAllocation.second>>>(d_memoryAllocation);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    return d_memoryAllocation;
}

__global__ void copyValue(float* d_target, float* d_valueToCopy, unsigned int size) {
    int ind = blockDim.x * blockIdx.x + threadIdx.x;

    if (ind < size) {
        d_target[ind] = d_valueToCopy[ind];
    }
}


// think about neccessety !!!!
// copies values of desire into unified memory, performs deepcopy
float* copyValuesUnified(float* d_valueToCopy, unsigned int size) {
    float* d_output;
    std::pair<unsigned int, unsigned int> blocksThread = computeBlockThreadAllocation(size);
    hipMallocManaged(&d_output, blocksThread.first * blocksThread.second * sizeof(float));

    copyValue<<<blocksThread.first, blocksThread.second>>>(d_output, d_valueToCopy, size);

    CHECK_CUDA_ERROR(hipGetLastError());

    return d_output;
}

float* copyValues(float* valueToCopy, int size) {
    // copies values of desire into device memory, performs deepcopy
    float* output;
    hipMalloc(&output, size * sizeof(float));

    int blockNum = size / 256;

    copyValue<<<blockNum + 1, BLOCK_SIZE>>>(output, valueToCopy, size);

    // do some error checking
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error while trying to copy values to cuda: %s\n", hipGetErrorString(error));
    }

    return output;
}

float* reserveMemoryOnDevice(unsigned int size) {
    // declare pointer
    float* memoryAlloc;

    // reserve actual space in memory, add some padding for thread efficiency
    CHECK_CUDA_ERROR(hipMalloc(&memoryAlloc, size + (size % BLOCK_SIZE)));

    // return pointer 
    return memoryAlloc;
}