#include "Tensor.h"
#include "cudaMem.cu"
#include "../autoDif/Function.h"


// only covers vectors and matrices
// prefix d_ marks values residing on gpu memory

    Tensor::Tensor(float* _value, unsigned int _shape_x, unsigned int _shape_y, bool _track_gradient, Function* _precedingFunction) {
        // shape_x is #rows and shape_y is #columns, 0 = no actual row/column

        // check for zero configuration
        if (_shape_x == 0) {
            throw std::runtime_error("Cannot initialize tensor that has 0 dimension");
        }
            
        this->d_value = _value;
        this->shape_x = _shape_x;
        this->shape_y = _shape_y;
        this->track_gradient = _track_gradient;
        this->precedingFunction = _precedingFunction;
        if (track_gradient) {
            d_gradient = reserveMemoryOnDevice(_shape_x * _shape_y);
        }
    }

    // basic functions

    float* Tensor::getValue() {
        return this->d_value;
    }

    float* Tensor::getGradient() {
        return this->d_gradient;
    }

    unsigned int Tensor::getShapeX() {
        return this->shape_x;
    }

    unsigned int Tensor::getShapeY() {
        return this->shape_y;
    }

    bool Tensor::getTrackGradient() {
        return this->track_gradient;
    }

    Function* Tensor::getPrecedingFunction() {
        return this->precedingFunction;
    }

    bool Tensor::sameShape(Tensor other) {
        // returns true if tensors are of same shape
        return (this->getShapeX() == other.getShapeX()) && (this->getShapeY() == other.getShapeY());
    }

    bool Tensor::matVecMulCompatible(Tensor other) {
        // returns true if thisTensor (matrix) x otherTensor (vector) can be performed
        return (this->getShapeY() == other.getShapeX()) && (other.getShapeY() == 0);
    }

    // operator overloading, Note: this class is specifically optimized for neural networks running on the gpu, therefore result of operation is stored in second tensor
    // => a + b updates value of b and returns pointer to tensor b

    Tensor* Tensor::operator+(Tensor &other) {
         // adds tensor values up and stores result in new Tensor, returns pointer to Tensor that holds result of addition if shapes match, otherwise prints error message and returns nullpointer
        // check if shapes match
        if (this->sameShape(other) && this->getShapeY() == 0) {
            /*CHECK_CUDA_ERROR(vecadd(this->getValue(), this->getShapeX(), other.getValue(), other.getShapeX(), other.getValue()));
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
            return &other;*/
        }
        // error handling if shapes do not match
        printf("Error: Tensors must have same shape and be vectors (shape_y = 0)\n");
        return nullptr;
    }

    Tensor* Tensor::operator-(Tensor &other) {
        return nullptr;
    }

    Tensor* Tensor::operator*(Tensor &other) {
        // Remark: only supports Matrix-Vector multiplication yet
        // Matrix-Vector multiplication
        if (matVecMulCompatible(other)) {
            
        }

        return nullptr;
    }

    Tensor* Tensor::operator%(Tensor &other) {
        // performs hadamard product
        return nullptr;
    }

    // activation functions
    // TODO

int main() {
    float h_bias[3] = {1.0f, 2.0f, 3.0f};
    float *bias;
    hipMalloc(&bias, 3 * sizeof(float));
    hipMemcpy(bias, h_bias, 3 * sizeof(float), hipMemcpyHostToDevice);

    Tensor t1 = Tensor(bias, 3, 0, false, nullptr);
    /*float *bias, *inp;
    hipError_t err;
    
    err = hipMalloc(&bias, 3 * sizeof(float));
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    err = hipMalloc(&inp, 3 * sizeof(float));
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        hipFree(bias);
        return 1;
    }

    // Initialize with some values
    float h_bias[3] = {1.0f, 2.0f, 3.0f};
    float h_inp[3] = {4.0f, 5.0f, 6.0f};
    
    hipMemcpy(bias, h_bias, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(inp, h_inp, 3 * sizeof(float), hipMemcpyHostToDevice);

    printf("Memory allocated and initialized\n");
    fflush(stdout);  // Force print

    Tensor t1 = Tensor(bias, 3, 0, false);
    Tensor t2 = Tensor(inp, 3, 0, false);
    
    printf("Tensors created\n");  // Debug print
    
    Tensor* t3 = t1 + t2;
    if (t3 == nullptr) {
        printf("Addition failed\n");
        hipFree(bias);
        hipFree(inp);
        return 1;
    }
    
    printf("Addition completed\n");  // Debug print
    
    float host_result[3];
    err = hipMemcpy(host_result, t3->getValue(), 3 * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("hipMemcpy failed: %s\n", hipGetErrorString(err));
        hipFree(bias);
        hipFree(inp);
        return 1;
    }
    
    printf("Results:\n");  // Debug print
    for (int i = 0; i < 3; i++) {
        printf("%f ", host_result[i]);
    }
    printf("\n");
    
    hipFree(bias);
    hipFree(inp);
    
    return 0;
    */
}