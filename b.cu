
#include <hip/hip_runtime.h>
#include <iostream>


hipError_t matvecmul(float* d_matix, unsigned int numRows, unsigned int numCols, float* d_vector, unsigned int vectorSize, float* d_targetMemorySpace) {
    if (numCols != vectorSize) {
        printf("vectors to be added have different shapes\n");
        return hipErrorInvalidValue;
    }
    
}


int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
           device, deviceProp.major, deviceProp.minor);
        std::cout << deviceProp.concurrentKernels;
    }
    return 0;
}